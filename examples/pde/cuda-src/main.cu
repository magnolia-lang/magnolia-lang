#include <vector>

#include <omp.h>

#include "gen/examples/pde/mg-src/pde-cuda.cuh"
#include "base.cuh"

//typedef array_ops::Shape Shape;
typedef array_ops<float>::Array Array;
typedef array_ops<float>::Index Index;
typedef array_ops<float>::Float Float;
typedef examples::pde::mg_src::pde_cuda::BasePDEProgram BasePDEProgram;
typedef examples::pde::mg_src::pde_cuda::PDEProgramDNF PDEProgramDNF;

void allocateDeviceMemory(Float* &u0_host_content, 
                          Float* &u1_host_content,    
                          Float* &u2_host_content,
                          Float* &u0_dev_content, 
                          Float* &u1_dev_content, 
                          Float* &u2_dev_content,
                          Array* &u0_dev, Array* &u1_dev, Array* &u2_dev) {

    hipMalloc((void**)&u0_dev_content, sizeof(Float) * TOTAL_PADDED_SIZE);
    hipMalloc((void**)&u1_dev_content, sizeof(Float) * TOTAL_PADDED_SIZE);
    hipMalloc((void**)&u2_dev_content, sizeof(Float) * TOTAL_PADDED_SIZE);

    hipMalloc((void**)&u0_dev, sizeof(*u0_dev));
    hipMalloc((void**)&u1_dev, sizeof(*u1_dev));
    hipMalloc((void**)&u2_dev, sizeof(*u2_dev));
}

void copyDeviceMemory(Float* &u0_host_content, 
                      Float* &u1_host_content,    
                      Float* &u2_host_content,
                      Float* &u0_dev_content, 
                      Float* &u1_dev_content, 
                      Float* &u2_dev_content,
                      Array* &u0_dev, Array* &u1_dev, Array* &u2_dev) {

    hipMemcpy(u0_dev_content, u0_host_content, sizeof(Float) * TOTAL_PADDED_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(u1_dev_content, u1_host_content, sizeof(Float) * TOTAL_PADDED_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(u2_dev_content, u2_host_content, sizeof(Float) * TOTAL_PADDED_SIZE, hipMemcpyHostToDevice);

    // Binding pointers with _dev
    hipMemcpy(&(u0_dev->content), &u0_dev_content, sizeof(u0_dev->content),    hipMemcpyHostToDevice);
    hipMemcpy(&(u1_dev->content), &u1_dev_content, sizeof(u1_dev->content),    hipMemcpyHostToDevice);
    hipMemcpy(&(u2_dev->content), &u2_dev_content, sizeof(u2_dev->content),    hipMemcpyHostToDevice);
}

int main() {
    size_t steps = 50;
    Array u0, u1, u2;
    dumpsine(u0);
    dumpsine(u1);
    dumpsine(u2);

    PDEProgramDNF pde_dnf;

    Float *u0_host_content, *u1_host_content, *u2_host_content;
    u0_host_content = u0.content;
    u1_host_content = u1.content;
    u2_host_content = u2.content;

    Float *u0_dev_content, *u1_dev_content, *u2_dev_content;
    Array *u0_dev, *u1_dev, *u2_dev;

    allocateDeviceMemory(u0_host_content, u1_host_content, u2_host_content,
        u0_dev_content, u1_dev_content, u2_dev_content,
        u0_dev, u1_dev, u2_dev);
    copyDeviceMemory(u0_host_content, u1_host_content, u2_host_content,
            u0_dev_content, u1_dev_content, u2_dev_content,
            u0_dev, u1_dev, u2_dev);
    
    float time;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    for (size_t i = 0; i < steps; ++i) {
        pde_dnf.step(*u0_dev, *u1_dev, *u2_dev);//, S_NU, S_DX, S_DT);
        std::cout << u0[PAD0 * PADDED_S1 * PADDED_S2 + PAD1 * PADDED_S2 + PAD2] << " "
                  << u1[PAD0 * PADDED_S1 * PADDED_S2 + PAD1 * PADDED_S2 + PAD2] << " "
                  << u2[PAD0 * PADDED_S1 * PADDED_S2 + PAD1 * PADDED_S2 + PAD2] << std::endl;
    }
    /*
    std::cout << end - begin << "[s] elapsed with sizes ("
              << S0 << ", "
              << S1 << ", "
              << S2 << ") with padding ("
              << PAD0 << ", "
              << PAD1 << ", "
              << PAD2 << ") on "
              << NB_CORES << " threads for "
              << steps << " steps" << std::endl;
              */
}
